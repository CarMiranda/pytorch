#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <cmath>
#include "fake_quantize_core.h"

/* FakeQuantize Op for PerChannelAffine quantization scheme */
namespace at {
namespace native {


/* Fake-quantizes the 'inputs' tensor.
Args:
  X: Forward input tensor.
  scale: scale of per tensor affine quantization
  zero_point: zero_point of per tensor affine quantization
  quant_min: minimum quantized value
  quant_max: maximum quantized value
  quant_delay: Count of global steps for which to delay the quantization.
               See note below.
  iter: The current quantization iteration used for `quant_delay`.
Returns:
  Quantized tensor (double dtype).

Notes:
  - quant_delay might be set to non-zero to help weights stabilize in the
    beginning of the training.
  - quantization range [quant_min, quant_max]
*/
Tensor fake_quantize_per_channel_affine_cuda(
    const Tensor& self,
    const Tensor& scale,
    const Tensor& zero_point,
    int64_t axis,
    int64_t quant_min,
    int64_t quant_max) {


  TORCH_CHECK(self.is_cuda());
  TORCH_CHECK(self.scalar_type() == ScalarType::Float);

  TORCH_CHECK(scale.size(0) == zero_point.size(0),
  "scale and zero-point need to have the same dimensions");
  TORCH_CHECK(scale.size(0) == self.size(axis),
  "dimensions of scale and zero-point are not consistent with input tensor")


  TORCH_CHECK(
      quant_min <= quant_max,
      "`quant_min` should be less than or \
        equal to `quant_max`.");


  TORCH_CHECK(at::min(zero_point).item().toLong() >= quant_min &&
              at::max(zero_point).item().toLong() <= quant_max,
      "`zero_point` must be between `quant_min` and `quant_max`.");

  TORCH_CHECK(axis >= 0 &&
              axis <= self.dim(),
      "`axis` must be between 0 and number of dimensions of input");

  auto Y = at::empty_like(self);
  for (int i = 0; i < self.size(axis); i++)
  {
    auto X_slice = self.slice(axis,i,i+1);
    auto Y_slice = Y.slice(axis,i,i+1);
    float sc = scale[i].item().toFloat();
    int64_t zp = zero_point[i].item().toLong();
    fake_quantize_slice_cuda(Y_slice, X_slice, sc, zp, quant_min, quant_max);
  }
  return Y;
}

/* Backward path to fake-quantize the 'inputs' tensor.

Args:
  X: Forward input tensor.
  dY: Backward input tensor.
  scale: scale of per tensor affine quantization
  zero_point: zero_point of per tensor affine quantization
  quant_min: minimum quantized value
  quant_max: maximum quantized value
  quant_delay: Count of global steps for which to delay the quantization.
               See note in forward.
  iter: The current quantization iteration used for `quant_delay`.
Returns:
  Quantized tensor (double dtype).

Notes:
  - quant_delay might be set to non-zero to help weights stabilize in the
    beginning of the training.
  - quantization range [quant_min, quant_max]
*/
Tensor fake_quantize_per_channel_affine_backward_cuda(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& scale,
    const Tensor& zero_point,
    int64_t axis,
    int64_t quant_min,
    int64_t quant_max) {

  TORCH_CHECK(dY.is_cuda());

  TORCH_CHECK(dY.scalar_type() == ScalarType::Float);
  TORCH_CHECK(X.scalar_type() == ScalarType::Float);

  TORCH_CHECK(X.numel() == dY.numel(), "`X` and `dY` are not the same size");
  TORCH_CHECK(
      quant_min <= quant_max,
      "`quant_min` should be less than or \
        equal to `quant_max`.");

  TORCH_CHECK(scale.size(0) == zero_point.size(0),
  "scale and zero-point need to have the same dimensions")
  TORCH_CHECK(scale.size(0) == X.size(axis),
  "dimensions of scale and zero-point are not consistent with input tensor")


  TORCH_CHECK(
      quant_min <= quant_max,
      "`quant_min` should be less than or \
        equal to `quant_max`.");


  TORCH_CHECK(at::min(zero_point).item().toLong() >= quant_min &&
              at::max(zero_point).item().toLong() <= quant_max,
      "`zero_point` must be between `quant_min` and `quant_max`.");

  TORCH_CHECK(axis >= 0 &&
              axis <= X.dim(),
      "`axis` must be between 0 and number of dimensions of input");


  if (X.numel() <= 0) {
    return X;
  }


  auto dX = dY.clone();
  for (int i = 0; i < X.size(axis); i++)
  {
    auto dY_slice = dY.slice(axis,i,i+1);
    auto X_slice = X.slice(axis,i,i+1);
    auto dX_slice = dX.slice(axis,i,i+1);
    float sc = scale[i].item().toFloat();
    int64_t zp = scale[i].item().toLong();
    fake_quantize_grad_slice_cuda(dX_slice, X_slice, dY_slice, sc, zp, quant_min, quant_max);
  }
  return dX;
}

} // namespace native
} // namespace at
